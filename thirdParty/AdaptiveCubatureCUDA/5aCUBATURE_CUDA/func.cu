#include "hip/hip_runtime.h"

DF func Fidx[2];

DF double f0(const double *y, int index, int id)
{  
  double g;
  g=1;
          for(int i=0;i<1;i++) 
  g*=exp(-y[i]);
 
  return g;

};

DF double f1(const double *y, int index, int id)
{  
  double g;
  g=1.0;
          for(int i=0;i<2;i++) 
  g*=exp(-y[i]);
 
  return g;

};



HDF double F(const double *y, int index,int id) 
{


  Fidx[1]=&f0;
  Fidx[0]=&f1;
  
  return Fidx[index](y,index,id);
};


